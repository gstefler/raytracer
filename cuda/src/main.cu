#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <math.h>
#include <float.h>
#include "vec3.cuh"
#include "ray.cuh"
#include "sphere.cuh"
#include "plane.cuh"
#include "scene.cuh"
#include "material.cuh"
#include "camera.cuh"
#include <hiprand/hiprand_kernel.h>

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

void writeImage(color *fb, int width, int height)
{
    std::ofstream out("image.ppm");
    out << "P3\n"
        << width << " " << height << "\n255\n";

    for (int j = 0; j < width * height; ++j)
    {
        out << static_cast<int>(255.999f * std::sqrt(fb[j].r())) << ' '
            << static_cast<int>(255.999f * std::sqrt(fb[j].g())) << ' '
            << static_cast<int>(255.999f * std::sqrt(fb[j].b())) << '\n';
    }
    out.close();
}

__device__ color ray_color(const ray &r, hitable *world, int depth, hiprandState *local_rand_state)
{
    ray cur_ray = r;
    color cur_attenuation(1.0f, 1.0f, 1.0f);

    for (int i = 0; i < depth; i++)
    {
        hit_record rec;
        if (world->hit(cur_ray, 0.001f, FLT_MAX, rec))
        {
            ray scattered;
            color attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec.normal, rec.p, attenuation, scattered, local_rand_state))
            {
                cur_attenuation = cur_attenuation * attenuation;
                cur_ray = scattered;
            }
            else
            {
                return color(0.0f, 0.0f, 0.0f);
            }
        }
        else
        {
            vec3 unit_direction = cur_ray.dir.normalized();
            float t = 0.5f * (unit_direction.y() + 1.0f);
            color c = (1.0f - t) * color(1.0f, 1.0f, 1.0f) + t * color(0.5f, 0.7f, 1.0f);
            return cur_attenuation * c;
        }
    }

    return color(0.0f, 0.0f, 0.0f);
}

__global__ void rand_init(hiprandState *rand_state, int width, int height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height))
        return;

    int pixel_index = j * width + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void create_scene(hitable **list, hitable **world, camera **cam, int width, int height)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        material *plane_mat = new material(color(0.5f, 0.5f, 0.5f), 1.0f);
        material *m1 = new material(color(0.1f, 0.2f, 0.5f), 1.0f);
        material *m2 = new material(color(0.01f, 0.01f, 0.01f), 0.05f);
        material *m3 = new material(color(1.0f, 1.0f, 1.0f), 0.0f);

        list[0] = new sphere(point3(-1.0f, 0.5f, -1.0f), 0.5f, m1);
        list[1] = new sphere(point3(0.0f, 0.5f, -1.5f), 0.5f, m3);
        list[2] = new sphere(point3(1.0f, 0.5f, -1.0f), 0.5f, m2);
        list[3] = new plane(point3(0.0f, 0.0f, 0.0f), vec3(0.0f, 1.0f, 0.0f), plane_mat);

        *world = new scene(list, 4);
        *cam = new camera(width, height);
    }
}

__global__ void render(color *fb, int width, int height, int samples, int max_depth,
                       hiprandState *rand_state, hitable **world, camera **cam)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height))
        return;

    int pixel_index = j * width + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    color pixel_color(0.0f, 0.0f, 0.0f);

    for (int s = 0; s < samples; s++)
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(width - 1);
        float v = (height - 1 - j + hiprand_uniform(&local_rand_state)) / float(height - 1);
        ray r = (*cam)->get_ray(u, v);
        pixel_color += ray_color(r, *world, max_depth, &local_rand_state);
    }

    pixel_color /= float(samples);
    fb[pixel_index] = pixel_color;
}

int main(void)
{
    int dev;
    checkCudaErrors(hipGetDevice(&dev));
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    std::cout << "Running on GPU: " << deviceProp.name << std::endl;

    int nx = 1920;
    int ny = 1080;
    int tx = 8;
    int ty = 8;
    int samples_per_pixel = 500;
    int max_depth = 10;

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMallocManaged((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));

    // Allocate scene objects and world
    hitable **d_list;
    checkCudaErrors(hipMallocManaged((void **)&d_list, 4 * sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMallocManaged((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMallocManaged((void **)&d_camera, sizeof(camera *)));

    // Initialize CUDA random state
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    rand_init<<<blocks, threads>>>(d_rand_state, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Create world
    create_scene<<<1, 1>>>(d_list, d_world, d_camera, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Setup CUDA events to measure render time
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start));
    // Render scene
    render<<<blocks, threads>>>(fb, nx, ny, samples_per_pixel, max_depth, d_rand_state, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));

    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Render time: " << milliseconds << " ms." << std::endl;

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    // Write image to file
    std::cout << "Rendering complete, writing image to file..." << std::endl;
    writeImage(fb, nx, ny);
    std::cout << "Image saved." << std::endl;

    // Cleanup
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    return 0;
}